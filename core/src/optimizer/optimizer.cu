#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////
// > Copyright (c) 2017 by Contributors. 
// > https://github.com/cjmcv
// > brief  
// > author Jianming Chen
////////////////////////////////////////////////////////////////

#include "util/device.h"
#include "optimizer/optimizer.h"

namespace dlex_cnn
{
	//SGD, w -= lr*g 
	template <typename Dtype>
	__global__ void sgd_update_kernel(const int n, const float lr, const Dtype* gradient, Dtype* weight)
	{
		CUDA_KERNEL_LOOP(index, n)
		{
			weight[index] -= lr * gradient[index];
		}
	}
	template <typename Dtype>
	void SGD<Dtype>::update_gpu(std::shared_ptr< Node<Dtype> > node)
	{
		const std::vector<std::shared_ptr<Tensor<Dtype>>> node_data = node->getDataVec();
		if (node_data.size() == 1)
			return;

		const std::vector<std::shared_ptr<Tensor<Dtype>>> op_data = node->getInteOp()->getOpGradient();

		Dtype* weight_data = (Dtype *)node_data[1]->getPushGpuData();
		const std::vector<int> weight_data_size = node_data[1]->getSize();
		const Dtype* w_gradient_data = (Dtype *)op_data[0]->getPushGpuData();
		int N = weight_data_size[tind::e4D];
		sgd_update_kernel<Dtype> << <DLEX_GET_BLOCKS(N), DLEX_CUDA_NUM_THREADS >> >(
			N, Optimizer<Dtype>::lr_,
			w_gradient_data, weight_data);

		if (node_data.size() >= 2 && op_data.size() >= 2)
		{
			Dtype* blas_data = (Dtype *)node_data[2]->getPushGpuData();
			const std::vector<int> blas_data_size = node_data[2]->getSize();
			const Dtype* b_gradient_data = (Dtype *)op_data[1]->getPushGpuData();
			N = blas_data_size[tind::e4D];
			sgd_update_kernel<Dtype> << <DLEX_GET_BLOCKS(N), DLEX_CUDA_NUM_THREADS >> >(
				N, Optimizer<Dtype>::lr_,
				b_gradient_data, blas_data);
		}
	}
	template void SGD<float>::update_gpu(std::shared_ptr< Node<float> > node);
	template void SGD<double>::update_gpu(std::shared_ptr< Node<double> > node);
}//namespace
